#include "hip/hip_runtime.h"
/*
  #include <cmath>
  #include <cstdio>

  #include <hip/hip_runtime.h>

  #include "dock.h"
  #include "gpu.cuh"
*/



__device__ void
InitRefMatrix_d (const int bidx, Ligand * __restrict__ mylig, const Protein * myprt)
{
  // lig loop, ~30
  for (int i = 0; i < lna_dc; i += blockDim.y) {
    const int l = i + threadIdx.y;
    if (l < lna_dc) {
      const int lig_t = mylig->t[l];

      // prt loop, ~300
      for (int j = 0; j < pnp_dc; j += blockDim.x) {
	const int p = j + threadIdx.x;
	if (p < pnp_dc) {
	  
	  const int prt_t = myprt->t[p];

	  const float dx = mylig->coord_new.x[l] - myprt->x[p];
	  const float dy = mylig->coord_new.y[l] - myprt->y[p];
	  const float dz = mylig->coord_new.z[l] - myprt->z[p];
	  const float dst = sqrtf (dx * dx + dy * dy + dz * dz);

	  const float pmf0 = enepara_dc->pmf0[lig_t][prt_t];
	  ref_matrix_dc->matrix[i][j] = (dst <= pmf0);
	}
      } // prt loop
    }
  } // lig loop
}






  

__device__ void
CalcMcc_d (const int bidx, Ligand * __restrict__ mylig, const Protein * myprt)
{
  // reduce
  __shared__ int tp[TperB];
  __shared__ int fn[TperB];
  __shared__ int fp[TperB];
  __shared__ int tn[TperB];
  tp[bidx] = 0;
  fn[bidx] = 0;
  fp[bidx] = 0;
  tn[bidx] = 0;
  __syncthreads ();

  // lig loop, ~30
  for (int i = 0; i < lna_dc; i += blockDim.y) {
    const int l = i + threadIdx.y;
    if (l < lna_dc) {
      const int lig_t = mylig->t[l];

      // prt loop, ~300
      for (int j = 0; j < pnp_dc; j += blockDim.x) {
	const int p = j + threadIdx.x;
	if (p < pnp_dc) {
	  
	  const int prt_t = myprt->t[p];

	  const float dx = mylig->coord_new.x[l] - myprt->x[p];
	  const float dy = mylig->coord_new.y[l] - myprt->y[p];
	  const float dz = mylig->coord_new.z[l] - myprt->z[p];
	  const float dst = sqrtf (dx * dx + dy * dy + dz * dz);
	  
	  const float pmf0 = enepara_dc->pmf0[lig_t][prt_t];
	  const int ref_val = ref_matrix_dc->matrix[i][j];
	  
	  tp[bidx] += (ref_val == 1 && dst <= pmf0);
	  fn[bidx] += (ref_val == 1 && dst > pmf0);
	  fp[bidx] += (ref_val == 0 && dst <= pmf0);
	  tn[bidx] += (ref_val == 0 && dst > pmf0);
	}
      } // prt loop
    }
  } // lig loop

  SumReduction_int_1D_4_d(bidx, tp, fn, fp, tn);
  
  if (bidx == 0) {
    const float tp0 = (float) tp[0];
    const float fn0 = (float) fn[0];
    const float fp0 = (float) fp[0];
    const float tn0 = (float) tn[0];
    const float dividend =  sqrtf ((tp0 + fp0) * (tp0 + fn0) * (tn0 + fp0) * (tn0 + fn0));

    if (dividend != 0)
      mylig->energy_new.cmcc = (tp0 * tn0 - fp0 * fn0) / dividend;
    else
      mylig->energy_new.cmcc = CMCC_INVALID_VAL;
      
      
  }

}
