#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define GD 16
#define BD 64
#define NT (BD * GD)
#define SZ 2000

# define CUDA_CALL ( x ) do { if (( x ) != hipSuccess ) { \
      printf (" Error at % s :% d \ n " , __FILE__ , __LINE__ ) ; \
      return EXIT_FAILURE ;}} while (0)

# define CURAND_CALL ( x ) do { if (( x ) != HIPRAND_STATUS_SUCCESS ) { \
      printf (" Error at % s :% d \ n " , __FILE__ , __LINE__ ) ; \
      return EXIT_FAILURE ;}} while (0)





__global__ void
kernel_init_state (hiprandState * state, int seed)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init (seed, tid, 0, &state[tid]);
  // seed, subsequence, offset, state
  // skipahead(100000,&state[tid]);
}


__global__ void
kernel_rand (float *table, hiprandState * state, int size)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int iter = (size + NT - 1) / NT;
  int iter_offset;
  float myrand;

  for (int i = 0; i < iter; i++) {
    iter_offset = NT * iter;
    if (tid < size - iter_offset) {
      myrand = hiprand_uniform (&state[tid]);	// range: [0,1]
      table[iter_offset + tid] = myrand;
    }
  }
}


void
check_table (float *table, int size)
{
  float item;
  for (int i = 0; i < size; i++) {
    item = table[i];
    printf ("%08d:\t%8.8f\n", i, item);
  }
}



void
run (int seed, int size)
{
  float *table, *table_dev;
  table = (float *) malloc (sizeof (float) * SZ);
  hipMalloc ((void **) &table_dev, sizeof (float) * SZ);

  hiprandState *state_dev;
  hipMalloc ((void **) &state_dev, sizeof (hiprandState) * NT);

  kernel_init_state <<< GD, BD >>> (state_dev, seed);
  kernel_rand <<< GD, BD >>> (table_dev, state_dev, size);

  hipMemcpy (table, table_dev, sizeof (float) * SZ, hipMemcpyDeviceToHost);

  check_table (table, size);

  free (table);
  hipFree (table_dev);
  hipFree (state_dev);
}


int
main (int argc, char **argv)
{
  srand (time(NULL));
  int seed = rand ();
  //int seed = 1234234;
  int size = SZ;

  if (argc == 2) {
    seed = atoi (argv[1]);
  }
  if (argc == 3) {
    seed = atoi (argv[1]);
    size = atoi (argv[2]);
  }

  run (seed, size);

  return 0;
}
